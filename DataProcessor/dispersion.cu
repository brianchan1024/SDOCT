#include "hip/hip_runtime.h"
#include "dispersion.cuh"

using namespace std;

Dispersion::Dispersion(Parameters &p, Data &d, Interpolation &i) : Helper(p), data(d), interp(i)
{
	cout << "Dispersion" << endl;
	d_fphi = NULL;

	dispParam.resize(4);

	dimGrid_w = dim3((width - 1) / TILE_WIDTH + 1, (height_1fr - 1)/TILE_WIDTH + 1, 1);
	dimGrid_w2 = dim3((width_2x - 1) / TILE_WIDTH + 1, (height_1fr - 1)/TILE_WIDTH + 1, 1);
	dimGrid_wt = dim3((width_trm - 1) / TILE_WIDTH + 1, (height_1fr - 1)/TILE_WIDTH + 1, 1);

	// half A-scan length parameters for ROI selection.
	width_h = p.numCameraPixels / 2;
	dimGrid_wh = dim3((width_h - 1) / TILE_WIDTH + 1, (height_1fr - 1)/TILE_WIDTH + 1, 1);
}

Dispersion::~Dispersion()
{
	if (d_fphi) gpuErrchk( hipFree(d_fphi) );
}

void Dispersion::process()
{
	gpuErrchk( hipMalloc((void **)&d_fphi, (p.batchAscans*width_2x*sizeof(float2))) );

	if (p.dispModeGS)
	{
		getROI();
		gridsearch();
		previewComp();
	}
	else
		dispModeMan();
}

void Dispersion::process(int dispModeGS, float &a2, float &a3)
{
	gpuErrchk( hipMalloc((void **)&d_fphi, (p.batchAscans*width_2x*sizeof(float2))) );

	if (dispModeGS)
	{
		getROI();
		gridsearch();
		previewComp();
		// return calculation result of dispersion params
		a2 = dispParam[0];
		a3 = dispParam[1];
	}
	else
	{
		dispModeMan();
	}

}

void Dispersion::dispModeMan()
{
	fphi = new float2[width_2x];
	vector<float> phi(width_2x, 0.0);
	
	dispParam[0] = p.dispA2;
	dispParam[1] = p.dispA3;
	dispParam[2] = dispParam[3] = 0;
	
	for (int i = 0; i < 4; i++)
		for (int j = 0; j < width_2x; j++)
			phi[j] = phi[j] + dispParam[i] * powf((j+1.f), (i+2));
	
	double temp = 0;
	for (int i = 0; i < width_2x; i++)
	{
		temp = -phi[i] / width_2x;
		fphi[i].x = cos(temp);
		fphi[i].y = sin(temp);
	}
	gpuErrchk( hipMemcpy(d_fphi, fphi, (width_2x * sizeof(float2)), hipMemcpyHostToDevice) );

	repmat<<<dimLine_w2,dimLine_B>>>(p.batchAscans, width_2x, d_fphi); gpuErrchk( hipPeekAtLastError() );
	delete[] fphi;
}

void Dispersion::getROI()
{
	float2 *initial_frame = new float2[height_1fr * width]();
	//data.loadFile(p.fnameData, 0, (height_1fr*width), initial_frame);
	data.loadFile(p.fnameBg, 0, (height_1fr*width), initial_frame);

	float2 *middle_frame = new float2[height_1fr * width]();
	data.loadFile(p.fnameData, (height_1fr * width_2x * (p.numBgFrame + p.dispFrame-1)), (height_1fr*width), middle_frame);

	cout << "	- Initialized dispesion parameters" << endl;
	cout << "file: " << p.fnameData << " height_1fr : " << height_1fr << " width: " << width << " width_2x: " << width_2x << " p.numBgFrame: " << p.numBgFrame << " p.dispFrame: " << p.dispFrame << endl;

	for(int i = 0; i < width_2x; i++)
	{
		cout << middle_frame[i].x << " ";
	}
	cout << endl;

	float *column_mean = new float[width];
	float column_mean_max = 0;
	columnMean(height_1fr, width, initial_frame, column_mean, column_mean_max);
	delete[] initial_frame;

	float *d_column_mean;
	float2 *d_middle_frame;
	float2 *d_IFT_data;
	gpuErrchk( hipMalloc((void **)&d_column_mean, (width * sizeof(float))) );
	gpuErrchk( hipMalloc((void **)&d_middle_frame, (height_1fr * width * sizeof(float2))) );
	gpuErrchk( hipMalloc((void **)&d_IFT_data, (height_1fr * width * sizeof(float2))) );
	gpuErrchk( hipMemcpy(d_middle_frame, middle_frame, (height_1fr *  width * sizeof(float2)), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(d_column_mean, column_mean, (width * sizeof(float)), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemset(d_IFT_data, 0, (height_1fr * width * sizeof(float2))) );
	delete[] middle_frame;
	delete[] column_mean;

	subtract<<<dimLine_w,dimLine_B>>>(height_1fr, width, d_middle_frame, d_column_mean, d_IFT_data); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_column_mean) );
	gpuErrchk( hipFree(d_middle_frame) );

	float2 *d_fringe_zp;
	gpuErrchk( hipMalloc((void **)&d_fringe_zp, (height_1fr * width_2x * sizeof(float2))) );
	gpuErrchk( hipMemset(d_fringe_zp, 0, (height_1fr * width_2x * sizeof(float2))) );  //memset for zeropad

	FFT(height_1fr, width, d_IFT_data, d_IFT_data);

	zero_pad<<<dimGrid_w,dimGrid_B>>>(height_1fr, width, d_IFT_data, d_fringe_zp); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_IFT_data) );

	IFT(height_1fr, width_2x, dimGrid_w2, dimGrid_B, d_fringe_zp, d_fringe_zp);
	
	gpuErrchk( hipMalloc((void **)&d_fringe_interp, (height_1fr * width_2x * sizeof(float2))) );
	gpuErrchk( hipMemset(d_fringe_interp, 0, (height_1fr * width_2x * sizeof(float2))) );
	
	interp.procInterp(height_1fr, width_2x, dimLine_w2, dimLine_B, d_fringe_zp, d_fringe_interp);
	gpuErrchk( hipFree(d_fringe_zp) );
	
	IFT(height_1fr, width_2x, dimGrid_w2, dimGrid_B, d_fringe_interp, d_fringe_interp);
		
	float2 *d_img_trim;
	gpuErrchk( hipMalloc((void **)&d_img_trim, (height_1fr * width_h * sizeof(float2))) );
	trim_width<<<dimGrid_wh,dimGrid_B>>>(height_1fr, width_2x, 1, width_h, d_fringe_interp, d_img_trim); gpuErrchk( hipPeekAtLastError() );

	float *d_img_mag;
	gpuErrchk( hipMalloc((void **)&d_img_mag, (height_1fr * width_h * sizeof(float))) );
	magnitude_db<<<dimGrid_wh,dimGrid_B>>>(height_1fr, width_h, d_img_trim, d_img_mag); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_img_trim) );

	float *d_img_transpose;
	gpuErrchk( hipMalloc((void **)&d_img_transpose, (width_h * height_1fr * sizeof(float))) );
	transpose<<<dimGrid_wh,dimGrid_B>>>(height_1fr, width_h, d_img_mag, d_img_transpose); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_img_mag) );

	float *TD = new float[width_h * height_1fr];
	gpuErrchk( hipMemcpy(TD, d_img_transpose, (width_h * height_1fr * sizeof(float)), hipMemcpyDeviceToHost) );
	gpuErrchk( hipFree(d_img_transpose) );

	Image selectROI(height_1fr, width_h, TD);
	selectROI.getPts();

	ROIbounds.resize(4);
	ROIbounds[0] = selectROI.ROIpts[0].x+1;  
	ROIbounds[1] = selectROI.ROIpts[0].y+1;		// Add plus one to these since rest of code was written
	ROIbounds[2] = selectROI.ROIpts[1].x+1;		// with matlab test values, which are 1 indexed.
	ROIbounds[3] = selectROI.ROIpts[1].y+1;		// Doesn't matter too much though.

	//ROIbounds[0] = 101;		// This is for gridsearch testing, if it's necessary to compare
	//ROIbounds[1] = 101,		// with matlab the same values can be entered into matlab as
	//ROIbounds[2] = 284,		// well (might need +/- 1). In general though, using manual
	//ROIbounds[3] = 222;		// mode and writing in a2/a3 in MATLAB is easier.

	delete[] TD;

	width_ROI = ROIbounds[3] - ROIbounds[1] + 1; //122
	height_ROI = ROIbounds[2] - ROIbounds[0] + 1; //184

	if ((width_ROI < 0) || (height_ROI < 0))
	{
		throw std::invalid_argument("ROI Error");
	}

	dimGrid_wf = dim3((width_2x - 1)/TILE_WIDTH + 1, (height_ROI - 1)/TILE_WIDTH + 1, 1);
	dimGrid_wROI = dim3((width_ROI - 1)/TILE_WIDTH + 1, (height_ROI - 1)/TILE_WIDTH + 1, 1);

	gpuErrchk( hipMalloc((void **)&d_fringe_frame, (height_ROI * width_2x * sizeof(float2))) );
	trim_height<<<dimGrid_w2,dimGrid_B>>>(height_1fr, width_2x, ROIbounds[0], ROIbounds[2], d_fringe_interp, d_fringe_frame); gpuErrchk( hipPeekAtLastError() );

	//hipFree(d_fringe_interp); // Keep these in memory for now
	//hipFree(d_fringe_frame);  // as they are used later.
}

void Dispersion::gridsearch()
{
	w.resize(width_2x);
	for (int i = 0; i < width_2x; i++)
		w[i] = i - width + 1;

	FFT(height_ROI, width_2x, d_fringe_frame, d_fringe_frame);
	
	fphi = new float2[width_2x];

	// prep a fft plan for getParam call inside secondOrder() and thirdOrder().
	int w2ROIh[2] = {width_2x, height_ROI};
	hipfftPlanMany(&plan_ROIw,1,w2ROIh,NULL,1,0,NULL,1,0,HIPFFT_C2C,height_ROI);

	secondOrder();
	thirdOrder();
	hipfftDestroy(plan_ROIw);
	
	genFphi();

	gpuErrchk( hipFree(d_fringe_frame) );
	delete[] fphi;

	cout << "	- Calculated gridsearch parameters" << endl;
}

void Dispersion::secondOrder()
{
	float temp = 0;
	a2.resize(p.dispTests);
	vector<float> R_a2(p.dispTests,0);
	linSpace(-0.3f, 0.3f, p.dispTests, a2);

	for (int i = 0; i < p.dispTests; i++)
	{
		for (int j = 0; j < width_2x; j++)
		{
			temp = (-a2[i] * powf(w[j],2)) / width_2x;
			fphi[j].x = cos(temp);
			fphi[j].y = sin(temp);
		}
		gpuErrchk( hipMemcpy(d_fphi, fphi, (width_2x * sizeof(float2)), hipMemcpyHostToDevice) );
		getParam(i, R_a2); // d_fphi is used in here
	}

	// Get the index of the min value.
	int na2 = 0;
    for (int i = 1; i < p.dispTests; i++)
        if(R_a2[i] < R_a2[na2])
            na2 = i;              
	dispParam[0] = a2[na2]; 

	cout << "	- Calculated a2:			" << dispParam[0] << endl;
}

void Dispersion::thirdOrder()
{
	float temp = 0;
	vector<float> a3(p.dispTests);
	linSpace(-0.1f, 0.1f, p.dispTests, a3);
	for (int i = 0; i < p.dispTests; i++)
		a3[i] /= width_2x;
	vector<float> R_a3(p.dispTests, 0);

	for (int i = 0; i < p.dispTests; i++)
	{
		for (int j = 0; j < width_2x; j++)
		{
			//temp =  dispParam[0] *  w[j] * w[j] + (a3[i] * w[j] * w[j] * w[j]);
			temp = (dispParam[0] * powf(w[j],2)) + (a3[i] * powf(w[j],3));
			temp = -temp / width_2x;
			fphi[j].x = cos(temp);
			fphi[j].y = sin(temp);
		}
		gpuErrchk( hipMemcpy(d_fphi, fphi, (width_2x * sizeof(float2)), hipMemcpyHostToDevice) );
		getParam(i, R_a3); // d_fphi is used in here
	}

	// Get the index of the min value.
	int na3 = 0;
    for (int i = 1; i < p.dispTests; i++)
        if(R_a3[i] < R_a3[na3])
            na3 = i;              

	dispParam[1] = a3[na3];
	dispParam[2] = dispParam[3] = 0;
	
	cout << "	- Calculated a3:			" << dispParam[1] << endl;
}

void Dispersion::genFphi()
{
	vector<float> phi(width_2x, 0.0);
	
	for (int i = 0; i < 4; i++)
		for (int j = 0; j < width_2x; j++)
			phi[j] = phi[j] + dispParam[i] * powf((j+1.f), (i+2));

	float temp = 0;
	for (int i = 0; i < width_2x; i++)
	{
		temp = -phi[i] / width_2x;
		fphi[i].x = cos(temp);
		fphi[i].y = sin(temp);
	}
	gpuErrchk( hipMemcpy(d_fphi, fphi, (width_2x * sizeof(float2)), hipMemcpyHostToDevice) );

	repmat<<<dimLine_w2,dimLine_B>>>(p.batchAscans, width_2x, d_fphi); gpuErrchk( hipPeekAtLastError() );
}

void Dispersion::previewComp()
{
	
	FFT(height_1fr, width_2x, d_fringe_interp, d_fringe_interp);
	
	float2 *d_comp;
	gpuErrchk( hipMalloc((void **)&d_comp, (height_1fr * width_2x * sizeof(float2))) );

	// matlab scales both real and imag compenets of fhi with the real component of S
	phi_multiply<<<dimLine_w2,dimLine_B>>>(height_1fr, width_2x, d_fphi, d_fringe_interp, d_comp); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_fringe_interp) );

	IFT(height_1fr, width_2x, dimGrid_w2, dimGrid_B, d_comp, d_comp);

	float2 *d_comp_trim;
	gpuErrchk( hipMalloc((void **)&d_comp_trim, (height_1fr * width_h * sizeof(float2))) );
	trim_width<<<dimGrid_w2,dimGrid_B>>>(height_1fr, width_2x, 1, width_h, d_comp, d_comp_trim); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_comp) );

	float *d_comp_mag;
	gpuErrchk( hipMalloc((void **)&d_comp_mag, (height_1fr * width_h * sizeof(float))) );
	magnitude_db<<<dimGrid_wh,dimGrid_B>>>(height_1fr, width_h, d_comp_trim, d_comp_mag); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_comp_trim) );

	float *d_comp_transpose;
	gpuErrchk( hipMalloc((void **)&d_comp_transpose, (width_h * height_1fr * sizeof(float))) );
	transpose<<<dimGrid_wh,dimGrid_B>>>(height_1fr, width_h, d_comp_mag, d_comp_transpose); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_comp_mag) );

	float *TD_comp_preview = new float[width_h * height_1fr];
	gpuErrchk( hipMemcpy(TD_comp_preview, d_comp_transpose, (width_h * height_1fr * sizeof(float)), hipMemcpyDeviceToHost) );
	gpuErrchk( hipFree(d_comp_transpose) );

	Image comp_image(height_1fr,width_h,TD_comp_preview);
	comp_image.dspl();

	delete[] TD_comp_preview;
	cout << "	- Generated compensated preview image" << endl;
}

void Dispersion::getParam(int it, vector<float> &result_vector)
{
	// matlab scales both real and imag compenets of fphi with the real component of S.
	float2 *d_S;
	gpuErrchk( hipMalloc((void **)&d_S, (height_ROI * width_2x * sizeof(float2))) );
	phi_multiply<<<dimLine_w2,dimLine_B>>>(height_ROI, width_2x, d_fphi, d_fringe_frame, d_S); gpuErrchk( hipPeekAtLastError() );

	cufftErrchk( hipfftExecC2C(plan_ROIw, d_S, d_S, HIPFFT_BACKWARD) );
	scale_IFT<<<dimGrid_w2,dimGrid_B>>>(height_ROI, width_2x, w2Recip, d_S);

	float2 *d_S_ROI;
	gpuErrchk( hipMalloc((void **)&d_S_ROI, (height_ROI * width_ROI * sizeof(float2))) );
	trim_width<<<dimGrid_wf,dimGrid_B>>>(height_ROI, width_2x, ROIbounds[1], ROIbounds[3], d_S, d_S_ROI); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_S) );

	float *d_S_abs;
	gpuErrchk( hipMalloc((void **)&d_S_abs, (height_ROI * width_ROI * sizeof(float))) );
	magnitude<<<dimGrid_wROI,dimGrid_B>>>(height_ROI, width_ROI, d_S_ROI, d_S_abs); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_S_ROI) );	

	float s = 0;
	sumElements(height_ROI, width_ROI, d_S_abs, s);
	divide<<<dimGrid_wROI,dimGrid_B>>>(height_ROI, width_ROI, d_S_abs, s, d_S_abs); gpuErrchk( hipPeekAtLastError() );

	float *d_S_log;
	gpuErrchk( hipMalloc((void **)&d_S_log, (height_ROI * width_ROI * sizeof(float))) );
	d_log<<<dimGrid_wROI,dimGrid_B>>>(height_ROI, width_ROI, d_S_abs, d_S_log); gpuErrchk( hipPeekAtLastError() );
	
	multiply<<<dimGrid_wROI,dimGrid_B>>>(height_ROI, width_ROI, d_S_abs, d_S_log, d_S_abs); gpuErrchk( hipPeekAtLastError() );
	sumElements(height_ROI, width_ROI, d_S_abs, result_vector[it]);
	gpuErrchk( hipFree(d_S_abs) );

	result_vector[it] = -result_vector[it];

	/*********************************************************************************************************************/
																											  /* Preview */
	float *d_S_transpose;
	gpuErrchk( hipMalloc((void **)&d_S_transpose, (width_ROI * height_ROI * sizeof(float))) );
	transpose<<<dimGrid_wROI,dimGrid_B>>>(height_ROI, width_ROI, d_S_log, d_S_transpose); gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(d_S_log) );

	float *compPrev = new float[width_ROI * height_ROI];
	gpuErrchk( hipMemcpy(compPrev, d_S_transpose, (width_ROI * height_ROI * sizeof(float)), hipMemcpyDeviceToHost) );
	gpuErrchk( hipFree(d_S_transpose) );

	Image dispCompImg(height_ROI, width_ROI, compPrev);
	dispCompImg.dsplGS();
	
	delete[] compPrev;
}

void Dispersion::linSpace(float min, float max, int pts, vector<float> &result_vector)
{
	float space = (max - min) / (pts - 1);
	result_vector[0] = min;
	
	for (int i = 1; i < pts; i++)
		result_vector[i] = result_vector[i - 1] + space;
}

void Dispersion::sumElements(int height, int width, float *initial_array, float &sum)
{	
		float *d_result_array;
		gpuErrchk( hipMalloc((void **)&d_result_array, (height * width * sizeof(float))) );
		gpuErrchk( hipMemcpy(d_result_array, initial_array, (height * width * sizeof(float)), hipMemcpyDeviceToDevice) );	
	
		float *d_temp_sum;
		gpuErrchk( hipMalloc((void **)&d_temp_sum, (height * width * sizeof(float))) );

		int sharedMemSize = TILE_WIDTH * TILE_WIDTH * sizeof(float);
		float sumHeight = height; 
		float sumWidth = width;
		dim3 dimGrid((width - 1)/TILE_WIDTH + 1, (height - 1)/TILE_WIDTH + 1, 1);
		dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
		int k = (height * width);

		while (k != 0)
		{
    		d_sum_elements<<<dimGrid, dimBlock, sharedMemSize>>>(sumHeight, sumWidth, d_result_array, d_temp_sum); gpuErrchk( hipPeekAtLastError() );
    		gpuErrchk( hipMemcpy(d_result_array, d_temp_sum, (sumHeight * sumWidth * sizeof(float)), hipMemcpyDeviceToDevice) );
   
    		sumHeight = ceil(sumHeight/TILE_WIDTH);
    		sumWidth = ceil(sumWidth/TILE_WIDTH);;
    		dimGrid.x = (sumWidth - 1) / TILE_WIDTH + 1; 
    		dimGrid.y = (sumHeight - 1) / TILE_WIDTH + 1;
    		k /= TILE_WIDTH*TILE_WIDTH;
		}

		gpuErrchk( hipMemcpy(&sum, &d_result_array[0], sizeof(float), hipMemcpyDeviceToHost) );

		gpuErrchk( hipFree(d_temp_sum) );
		gpuErrchk( hipFree(d_result_array) );
}

vector<float> Dispersion::getDispersionCalResult()
{
	return dispParam;
}