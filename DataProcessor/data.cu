#include "hip/hip_runtime.h"
#include "data.cuh"
#include <string>
#include <ctime>
//#include "DP_logger.h"

using namespace std;

Data::Data(Parameters &p) : Helper(p)
{
	d_gauss_win = NULL;
	genGaussWin();
}

Data::~Data()
{
	if (d_gauss_win) gpuErrchk( hipFree(d_gauss_win) );
}

void Data::genGaussWin()
{
	float temp;
	float *gauss_win = new float[width_2x];
    for (int i = 0; i < width_2x; i++)
	{
        temp = p.alpha * (((i+1.f) / width) - 1.f);
		temp *= temp;
        gauss_win[i] = expf(temp*(-0.5f));
    }

	gpuErrchk( hipMalloc((void **)&d_gauss_win, (p.batchAscans * width_2x * sizeof(float))) );
	gpuErrchk( hipMemcpy(d_gauss_win, gauss_win, (width_2x * sizeof(float)), hipMemcpyHostToDevice) );
	repmat<<<dimLine_w2,dimLine_B>>>(p.batchAscans, width_2x, d_gauss_win); gpuErrchk( hipPeekAtLastError() );

	delete[] gauss_win;

	cout <<"	- Gauss win matrix created" << endl;
}

void Data::loadFile(string fname, int length, float *result_array)
{
	std::time_t rawtime;
    std::tm* timeinfo;
    char buffer [80];
    std::time(&rawtime);
    timeinfo = std::localtime(&rawtime);
    std::strftime(buffer,80,"%Y-%m-%d-%H-%M-%S",timeinfo);
	cout << __FUNCTION__ << "before load file 1" << fname << "\t" << buffer << endl;

    ifstream is(fname);
	if (is)
	{
		for (int i = 0; i < length; i++)
			is >> result_array[i];
		is.close();
	}
	else
	{
		cerr << "Calibration File could not be opened!\n"; // Report error
		cerr << "Error code: " << strerror(errno); // Get some info as to why
		//system("pause");
		//DP_LOGGER_ERROR << __FUNCTION__ << " Calibration File could not be opened error.";
		throw invalid_argument("Calibration File Error");
	}

	std::time(&rawtime);
    timeinfo = std::localtime(&rawtime);
    std::strftime(buffer,80,"%Y-%m-%d-%H-%M-%S",timeinfo);
	cout << __FUNCTION__ << "after load file 1" << fname << "\t" << buffer << endl;
}

// for background and disperion
void Data::loadFile(string fname, int startLocation, int length, float2 *result_array)
{
	std::time_t rawtime;
    std::tm* timeinfo;
    char buffer [80];
    std::time(&rawtime);
    timeinfo = std::localtime(&rawtime);
    std::strftime(buffer,80,"%Y-%m-%d-%H-%M-%S",timeinfo);
	cout << "**********"<<endl;
	cout << __FUNCTION__ << " before load file 2" << fname << "\t" << buffer << endl;

	cout << fname << " startLocation: " << startLocation << endl;
	ifstream is(fname, ios::in|ios::binary);
	if (is)
	{
		is.seekg(startLocation,ios::beg);
	
		int x = 0;
		for (int i = 0; i < length; ++i)
		{
			is.read(reinterpret_cast<char*>(&x), sizeof(uint16_t));
			result_array[i].x = (float)x;
			//result_array[i].y = 0;

			if(i < 2048)
			{
				cout<< x << " ";
			}
		}
		cout << endl;
		is.close();
	}
	else
	{
		
		cerr << " File could not be opened!\n"; // Report error
		cerr << "Error code: " << strerror(errno) << "\n"; // Get some info as to why
		//system("pause");
		//DP_LOGGER_ERROR << __FUNCTION__ << " File could not be opened error.";
		throw invalid_argument("File Open Error");
	}

	cout << "in load file 2 \n";
	for(int i = 0; i < width_2x; i++)
	{
		cout << result_array[i].x << " ";
	}
	cout << endl;

	std::time(&rawtime);
    timeinfo = std::localtime(&rawtime);
    std::strftime(buffer,80,"%Y-%m-%d-%H-%M-%S",timeinfo);
	cout << __FUNCTION__ << " after load file 2" << fname << "\t" << buffer << endl;
	cout << "**********"<<endl<<endl<<endl;
}

void Data::loadFile(string fname, int startLocation, int length, uint16_t *result_array)
{
	std::time_t rawtime;
    std::tm* timeinfo;
    char buffer [80];
    std::time(&rawtime);
    timeinfo = std::localtime(&rawtime);
    std::strftime(buffer,80,"%Y-%m-%d-%H-%M-%S",timeinfo);
	cout << __FUNCTION__ << " before load file 3" << fname << "\t" << buffer;

	cout << fname << endl;
	length *=2;
	ifstream is(fname, ios::in|ios::binary);
	if (is.is_open())
	{
		is.seekg (startLocation, ios::beg);
		is.read(reinterpret_cast<char*>(result_array), length);
		is.close();
	}
	else
	{
		cerr << "File could not be opened!\n";
		cerr << "Error code: " << strerror(errno);
		//system("pause");
		//DP_LOGGER_ERROR << __FUNCTION__ << " Data File Error.";
		throw invalid_argument("Data File Error");
	}

	std::time(&rawtime);
    timeinfo = std::localtime(&rawtime);
    std::strftime(buffer,80,"%Y-%m-%d-%H-%M-%S",timeinfo);
	cout << __FUNCTION__ << " after load file 3" << fname << "\t" << buffer;
}