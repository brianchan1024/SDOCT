#pragma once

//#include <stdio.h>

#include "parameters.h"
#include "data.cuh"
#include "interpolation.cuh"
#include "dispersion.cuh"
#include "background.cuh"
#include "output.cuh"

//using namespace std;

// add file progress output
// test height for each class.


int main()
{
	Parameters *p = new Parameters();
	Data *data = new Data(*p);
	Interpolation *interp = new Interpolation(*p, *data);
	interp->init();

	Dispersion *disp = new Dispersion(*p, *data, *interp);
	disp->process();

	Background *bg = new Background(*p, *data, *interp, *disp);
	bg->process();

	Output *out = new Output(*p, *data, *interp, *disp, *bg);
	out->process("test");

	delete out;
	delete bg;
	delete disp;
	delete interp;
	delete data;
	delete p;

	hipDeviceReset();
	return 0;
}