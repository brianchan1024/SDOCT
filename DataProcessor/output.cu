#include "hip/hip_runtime.h"
#include "output.cuh"
//#include "hip/hip_runtime_api.h"

using namespace std;
using namespace cv;

Output::Output(Parameters &p, Data &da, Interpolation &i, Dispersion &di, Background &b) : Helper(p), data(da), interp(i), disp(di), bg(b)
{
	cout << "****************\nData processing" << endl;
	
	height_ba = p.numAscansPerBscan/p.batchAscans;
	width_ba = p.numCameraPixels*p.batchAscans;
	width_2xba = (p.numCameraPixels*2)*p.batchAscans;
	frames_tot = p.numBScans*p.batchFrames;
	//std::cout<<"Debug: "<<__FUNCTION__<<" frames_tot:"<<frames_tot<<" p.numBScans: "<<p.numBScans<<" p.batchFrames: "<<p.batchFrames<<std::endl;
	dimGrid_w = dim3((width - 1) / TILE_WIDTH + 1, (height_bfr - 1)/TILE_WIDTH + 1, 1);
	dimGrid_w2 = dim3((width_2x - 1) / TILE_WIDTH + 1, (height_bfr - 1)/TILE_WIDTH + 1, 1);
	dimGrid_wt = dim3((width_trm - 1) / TILE_WIDTH + 1, (height_bfr - 1)/TILE_WIDTH + 1, 1);

	dimLine_wba	= dim3((width_ba+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, 1, 1);
	dimLine_w2xba = dim3((width_2xba+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, 1, 1);
	dimLine_wtba = dim3(((width_trm*p.batchAscans)+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, 1, 1);
}

Output::~Output()
{
}

void Output::process(std::string datatype)
{
	hipMemGetInfo(&mem_avail, &mem_total);
	std::cout<<"Debug: "<<__FUNCTION__<<"0 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	cv::Mat prevImg;

	// Hold the full processed image stack in memory.
	float *processed_data_array = new float[width_trm * height_bfr * frames];
	initResources();
	//std::cout<<"Debug: "<<__FUNCTION__<<" process 1"<<std::endl;
	//gpuErrchk( hipProfilerStart() );
	for (int i = 0; i < frames; i++)
	{	
		processData(i, processed_data_array);
	
		/*
		// do not need to show the processed image, since we have a preview UI @brian
		if (i%p.prevFreq == 0)
		{
			prevImg = cv::Mat(width_trm, height_1fr, CV_32F, &processed_data_array[i * width_trm * height_bfr]);
			cv::namedWindow("Preview", cv::WINDOW_AUTOSIZE);
			cv::imshow("Preview", prevImg);
			cv::waitKey(1);
		}
		*/
	}
	//std::cout<<"Debug: "<<__FUNCTION__<<" process 2"<<std::endl;
	//destroyWindow("Preview"); @brian
	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"1 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	freeResources();
	//gpuErrchk( hipProfilerStop() );

	writeToDisk(processed_data_array, datatype);

	delete[] processed_data_array;
	prevImg.release(); 
	//std::cout<<"Debug: "<<__FUNCTION__<<" process 2"<<std::endl;
	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"2 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;
}

void Output::initResources()
{
	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"0 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;
	//// debug
	//output_csv(p.batchAscans, width_2x, data.d_gauss_win, "gaussWin.csv");
	//output_csv(p.batchAscans,width_2x,interp.d_query_points,"pixel_nonuniform.csv");  // make this private
	//output_csv(p.batchAscans,width_2x,disp.d_fphi,"fphi_imag.csv");
	//output_csv(p.batchAscans,width,bg.d_bg,"bg.csv");
	//output_csv(p.batchAscans,width_2x,bg.d_bg_mask,"bg_mask.csv");
	//output_csv(p.batchAscans,width_trm,bg.d_bg_noise,"bg_noise.csv");
	//if(it==3) output_csv((width_trm*4),(height_bfr/4),d_proc_buff_trns,"4frames.csv");	// scaling might be wrong.
	//if(it==12) output_csv(width_trm,height_bfr,d_proc_buff_trns,"1frames.csv");			// these two need to be moved to process loop if used.

	h_buff_1 = new uint16_t[height_bfr * width]();

	gpuErrchk( hipHostRegister(h_buff_1, (height_bfr*width*sizeof(uint16_t)), hipHostRegisterPortable));
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipStreamCreate(&stream1) );
	gpuErrchk( hipStreamCreate(&stream2) );

	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"1 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	int m[2] = {width, height_bfr};
	// mem leak here
	cufftErrchk( hipfftPlanMany(&plan_w,1,m,NULL,1,0,NULL,1,0,HIPFFT_C2C,height_bfr) );

	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"2 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	// mem leak here
	cufftErrchk( hipfftSetStream(plan_w,stream1) );

	int n[2] = {width_2x, height_bfr};
	cufftErrchk( hipfftPlanMany(&plan_w2,1,n,NULL,1,0,NULL,1,0,HIPFFT_C2C,height_bfr) );

	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"3 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;
	
	cufftErrchk( hipfftSetStream(plan_w2,stream1) );
	
	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"4 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	// error here betwwn nezt debug info
	gpuErrchk( hipMalloc((void **)&d_raw_data, (height_bfr * width * sizeof(uint16_t))) );

	//hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"3 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipMalloc((void **)&d_proc_buff_0, (height_bfr * width * sizeof(float2))) );

	//hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"4 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipMalloc((void **)&d_proc_buff_1, (height_bfr * width_2x * sizeof(float2))) );

	//hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"5 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	// error occurs here
	//hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"6 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipMalloc((void **)&d_proc_buff_2, (height_bfr * width_2x * sizeof(float2))) );

	//hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"7 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipMalloc((void **)&d_proc_buff_trm, (height_bfr * width_trm * sizeof(float2))) );

	//hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"8 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipMalloc((void **)&d_proc_buff_db, (height_bfr * width_trm * sizeof(float))) );

	//hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"8.5 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipMalloc((void **)&d_proc_buff_trns, (width_trm * height_bfr * sizeof(float))) );
	
	//hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"9 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	data.loadFile(p.fnameData, (height_1fr * width_2x * p.numBgFrame), (height_bfr*width), h_buff_1);

	//hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"10 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipMemcpy(d_raw_data, h_buff_1, (height_bfr * width * sizeof(uint16_t)), hipMemcpyHostToDevice) );
	
	//hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"11 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;
}

void Output::processData(int it, float *proc_data_piece)
{
	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"0 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipMemset(d_proc_buff_0, 0, (height_bfr * width * sizeof(float2))) );
	subtract<<<dimLine_wba,dimLine_B,0,stream1>>>(height_ba, width_ba, d_raw_data, bg.d_bg, d_proc_buff_0); gpuErrchk( hipPeekAtLastError() );
	
	cufftErrchk( hipfftExecC2C(plan_w, d_proc_buff_0, d_proc_buff_0, HIPFFT_FORWARD) );
	gpuErrchk( hipMemset(d_proc_buff_1, 0, (height_bfr * width_2x * sizeof(float2))) );
	
	zero_pad<<<dimGrid_w,dimGrid_B,0,stream1>>>(height_bfr, width, d_proc_buff_0, d_proc_buff_1); gpuErrchk( hipPeekAtLastError() );

	cufftErrchk( hipfftExecC2C(plan_w2, d_proc_buff_1, d_proc_buff_1, HIPFFT_BACKWARD) );
	scale_IFT_x<<<dimGrid_w2,dimGrid_B,0,stream1>>>(height_bfr, width_2x, w2Recip, d_proc_buff_1); gpuErrchk( hipPeekAtLastError() );

	interp.procInterp(height_ba, width_2xba, dimLine_w2xba, dimLine_B, stream1, d_proc_buff_1, d_proc_buff_2);

	data.loadFile(p.fnameData, ((height_1fr * width_2x) * (p.numBgFrame+((it+1)*p.batchFrames))), (height_bfr*width), h_buff_1);
	
	//std::cout<<"Debug: "<<__FUNCTION__<<" processData 1"<<std::endl;

	mult_divide<<<dimLine_w2xba,dimLine_B,0,stream1>>>(height_ba, width_2xba, data.d_gauss_win, d_proc_buff_2, bg.d_bg_mask, d_proc_buff_2); gpuErrchk( hipPeekAtLastError() );
	phi_multiply<<<dimLine_w2xba,dimLine_B,0,stream1>>>(height_ba, width_2xba, disp.d_fphi, d_proc_buff_2, d_proc_buff_1); gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpyAsync(d_raw_data, h_buff_1, (height_bfr * width * sizeof(uint16_t)), hipMemcpyHostToDevice, stream2) );

	cufftErrchk( hipfftExecC2C(plan_w2,d_proc_buff_1, d_proc_buff_1, HIPFFT_BACKWARD) );
	scale_IFT<<<dimGrid_w2,dimGrid_B,0,stream1>>>(height_bfr, width_2x, w2Recip, d_proc_buff_1); gpuErrchk( hipPeekAtLastError() );

	trim_width<<<dimGrid_wt,dimGrid_B,0,stream1>>>(height_bfr, width_2x, p.startPixel, p.endPixel, d_proc_buff_1, d_proc_buff_trm); gpuErrchk( hipPeekAtLastError() );
	magnitude_db<<<dimGrid_wt,dimGrid_B,0,stream1>>>(height_bfr, width_trm, d_proc_buff_trm, d_proc_buff_db); gpuErrchk( hipPeekAtLastError() );
	subt_divide<<<dimLine_wtba,dimLine_B,0,stream1>>>(height_ba, (width_trm*p.batchAscans), d_proc_buff_db, bg.d_bg_noise, grayRecip, d_proc_buff_db); gpuErrchk( hipPeekAtLastError() );
	multiframe_transpose<<<dimGrid_wt,dimGrid_B,0,stream1>>>(height_1fr, width_trm, p.batchFrames, d_proc_buff_db, d_proc_buff_trns); gpuErrchk( hipPeekAtLastError() );
	
	int write_offset = (it*height_bfr*width_trm);
	gpuErrchk( hipMemcpy((proc_data_piece+write_offset), d_proc_buff_trns, (width_trm * height_bfr * sizeof(float)), hipMemcpyDeviceToHost) );

	gpuErrchk( hipDeviceSynchronize() );
	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"0 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

}

void Output::writeToDisk(float *proc_data_array, std::string datatype)
{
	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"0 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	Mat image;
	Mat new_image;
	char fname_opencv[200];
	string path_string; 

	Mat row_mean;
	Mat avg_img;
	
	if (!p.reslice)
	{
		std::cout<<"Debug: "<<__FUNCTION__<<"frames_tot:"<<frames_tot<<std::endl;

		unsigned slash_split = p.fnameData.find_last_of("/\\");
		string path_name = p.fnameData.substr(0,slash_split);
		string f_name = p.fnameData.substr(slash_split+1);
		unsigned period_split = f_name.find_last_of("."); 
		string r_name =  f_name.substr(0, period_split);
		//path_string = (path_name+"\\"+r_name);
		path_string = (path_name+"\\GPU_processed");
		CreateDirectory(path_string.c_str(),NULL);
		path_string = (path_string + "\\" + r_name);
		CreateDirectory(path_string.c_str(),NULL);

		if (datatype == "fundus")
		{	
			sprintf(fname_opencv,"%s\\0.png",path_string.c_str());
			// fundus view result in one image
			for (int i = 0; i < frames_tot; i++)
			{
				image = cv::Mat(width_trm, height_1fr, CV_32F, &proc_data_array[i*width_trm*height_1fr]);
				new_image = cv::Mat(width_trm, height_1fr, CV_8U);
				image.convertTo(new_image,CV_8U,255);

				reduce(new_image,row_mean, 0, CV_REDUCE_AVG);
				avg_img.push_back(row_mean);
			}
			imwrite(fname_opencv, avg_img);
		}
		else {
			// process the other data type
			for (int i = 0; i < frames_tot; i++)
			{
				// This conversion makes the images look like they do in MATLAB. Doing normalization, like in the
				// image class, results on low contrast images. This needs to be confirmed, but I believe all values below 0 are set to 0.
				image = cv::Mat(width_trm, height_1fr, CV_32F, &proc_data_array[i*width_trm*height_1fr]);
				new_image = cv::Mat(width_trm, height_1fr, CV_8U);
				image.convertTo(new_image,CV_8U,255);
				//new_image = cv::Mat(width_trm, height_1fr, CV_16U);
				//image.convertTo(new_image,CV_16U,65535);	

				reduce(new_image,row_mean, 0, CV_REDUCE_AVG);
				avg_img.push_back(row_mean);
				// Try to create the directory on the first iteration.
				//std::cout<<"Debug: "<<__FUNCTION__<<"path_string:"<<path_string<<std::endl;
				//sprintf(fname_opencv,"%s\\%i.tiff",path_string.c_str(),i);
				sprintf(fname_opencv,"%s\\%i.png",path_string.c_str(),i);
				imwrite(fname_opencv, new_image);
			}
		}
		/*
		for (int i = 0; i < frames_tot; i++)
		{

			// This conversion makes the images look like they do in MATLAB. Doing normalization, like in the
			// image class, results on low contrast images. This needs to be confirmed, but I believe all values below
			// 0 are set to 0.
			image = cv::Mat(width_trm, height_1fr, CV_32F, &proc_data_array[i*width_trm*height_1fr]);
			new_image = cv::Mat(width_trm, height_1fr, CV_8U);
			image.convertTo(new_image,CV_8U,255);
			//new_image = cv::Mat(width_trm, height_1fr, CV_16U);
			//image.convertTo(new_image,CV_16U,65535);	

			reduce(new_image,row_mean, 0, CV_REDUCE_AVG);
			avg_img.push_back(row_mean);
			// Try to create the directory on the first iteration.
			if (i == 0) 
			{
				unsigned slash_split = p.fnameData.find_last_of("/\\");
				string path_name = p.fnameData.substr(0,slash_split);
				string f_name = p.fnameData.substr(slash_split+1);
				unsigned period_split = f_name.find_last_of("."); 
				string r_name =  f_name.substr(0, period_split);
				//path_string = (path_name+"\\"+r_name);
				path_string = (path_name+"\\GPU_processed");
				CreateDirectory(path_string.c_str(),NULL);
				path_string = (path_string + "\\" + r_name);
				CreateDirectory(path_string.c_str(),NULL);
			}
			//std::cout<<"Debug: "<<__FUNCTION__<<"path_string:"<<path_string<<std::endl;
			//sprintf(fname_opencv,"%s\\%i.tiff",path_string.c_str(),i);
			sprintf(fname_opencv,"%s\\%i.png",path_string.c_str(),i);
			imwrite(fname_opencv, new_image);
		}
		imwrite("E:\\a.png", avg_img);
		*/
	}

	// reslice before saving to disk
	else
	{
		float *resliced_data_array = new float[height_bfr*width_trm*frames];

		for (int i = 0; i < width_trm; i++)
			for (int j = 0; j < frames_tot; j++) 
				for (int k = 0; k < height_1fr; k++)
					resliced_data_array[i*frames_tot*height_1fr+j*height_1fr+k] = 
						proc_data_array[j*width_trm*height_1fr+i*height_1fr+k];

		for (int i = 0; i < width_trm; i ++)	
		{
			image = Mat(frames_tot, height_1fr, CV_32F, &resliced_data_array[i*frames_tot*height_1fr]);
			new_image = Mat(frames_tot, height_1fr, CV_8U);
			image.convertTo(new_image, CV_8U, 255);
			//new_image = cv::Mat(width_trm, height_1fr, CV_16U);
			//image.convertTo(new_image,CV_16U,65535);

			cv::namedWindow("Preview", cv::WINDOW_AUTOSIZE);
			cv::imshow("Preview", new_image);			
			cv::waitKey(1);
			
			// Try to create the directory on the first iteration.
			if (i == 0)
			{
				unsigned slash_split = p.fnameData.find_last_of("/\\");
				string path_name = p.fnameData.substr(0,slash_split);
				string f_name = p.fnameData.substr(slash_split+1);
				unsigned period_split = f_name.find_last_of("."); 
				string r_name =  f_name.substr(0, period_split);
				//string path_string = (path_name+"\\"+r_name+" - reslice");

				path_string = (path_name+"\\GPU_processed");
				CreateDirectory(path_string.c_str(),NULL);
				path_string = (path_string + "\\" + r_name + " - reslice");
				CreateDirectory(path_string.c_str(),NULL);
			}

			sprintf(fname_opencv,"%s\\%i.tiff",path_string.c_str(),i);
			imwrite(fname_opencv, new_image);
		}
		destroyWindow("Preview");
		delete[] resliced_data_array;
	}

	
	image.release();
	new_image.release();
	cv::waitKey(1);
	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"1 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;
}

void Output::freeResources()
{
	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"-1 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	// modify here for cuda error @brian
	hipfftDestroy(plan_w);
	hipfftDestroy(plan_w2);

	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"0 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipStreamDestroy(stream1) );
	gpuErrchk( hipStreamDestroy(stream2) );

	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"1 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipFree(d_raw_data) );
	gpuErrchk( hipFree(d_proc_buff_0) );
	gpuErrchk( hipFree(d_proc_buff_trm) );
	gpuErrchk( hipFree(d_proc_buff_db) );
	gpuErrchk( hipFree(d_proc_buff_trns) );
	gpuErrchk( hipFree(d_proc_buff_1) );
	gpuErrchk( hipFree(d_proc_buff_2) );

	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"2 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;

	gpuErrchk( hipHostUnregister(h_buff_1) );
	delete[] h_buff_1;

	hipMemGetInfo(&mem_avail, &mem_total);
	//std::cout<<"Debug: "<<__FUNCTION__<<"3 mem avail: "<<mem_avail<<" total: "<<mem_total<<std::endl;
}